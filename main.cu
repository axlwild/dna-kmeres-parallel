#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <set>
#include <math.h>
#include <algorithm>
#include <sstream>
#include <typeinfo>
#include "hip/hip_runtime.h"

using namespace std;

int numberOfSequenses = 0;



// Method definition
void importSeqs(string inputFile);
void printSeqs();
void getPermutations(char *str, char* permutations, int last, int index);
int permutationsCount(string permutation, string sequence, int k);
void sequentialKmerCount(vector<string> &seqs, vector<string> &permutations , int k);
void doSequentialKmereDistance();
// Vectors to store ids and seqs
vector<string> ids;
vector<string> seqs;
vector<int> indexes_aux;

// Device variables.
char * d_data; // all the strings.
unsigned int * d_indices;
float * d_distances;
const char * data;

// coincidences of k-mer on each input
int * d_sums;
int * h_sums;

// number of permutations of RNA K_meres and k-value
__constant__ char c_perms[64][4] = {
    "AAA", "AAC", "AAT","AAG",
    "ACA", "ACC", "ACT","ACG",
    "ATA", "ATC", "ATT","ATG",
    "AGA", "AGC", "AGT","AGG",

    "CAA", "CAC", "CAT","CAG",
    "CCA", "CCC", "CCT","CCG",
    "CTA", "CTC", "CTT","CTG",
    "CGA", "CGC", "CGT","CGG",

    "GAA", "GAC", "GAT","GAG",
    "GCA", "GCC", "GCT","GCG",
    "GTA", "GTC", "GTT","GTG",
    "GGA", "GGC", "GGT","GGG",

    "TAA", "TAC", "TAT","TAG",
    "TCA", "TCC", "TCT","TCG",
    "TTA", "TTC", "TTT","TTG",
    "TGA", "TGC", "TGT","TGG",
};
__constant__ int  c_size ;
char perms[64][4] = {
                        "AAA", "AAC", "AAT","AAG",
                        "ACA", "ACC", "ACT","ACG",
                        "ATA", "ATC", "ATT","ATG",
                        "AGA", "AGC", "AGT","AGG",

                        "CAA", "CAC", "CAT","CAG",
                        "CCA", "CCC", "CCT","CCG",
                        "CTA", "CTC", "CTT","CTG",
                        "CGA", "CGC", "CGT","CGG",

                        "GAA", "GAC", "GAT","GAG",
                        "GCA", "GCC", "GCT","GCG",
                        "GTA", "GTC", "GTT","GTG",
                        "GGA", "GGC", "GGT","GGG",

                        "TAA", "TAC", "TAT","TAG",
                        "TCA", "TCC", "TCT","TCG",
                        "TTA", "TTC", "TTT","TTG",
                        "TGA", "TGC", "TGT","TGG",
};
int permsSize = sizeof(perms) ;

vector<string> permutationsList (perms, end(perms));

float ** distancesSequential;
float ** distancesParallel;
/*
string join(const vector<string>& vec, const char* delim){
    stringstream res;
    copy(vec.begin(), vec.end(), ostream_iterator<string>(res, delimiter.c_str()));
    return res.str();
}
 */
string join(const std::vector<std::string> &lst, const std::string &delim){
    std::string ret;
    for(const auto &s : lst) {
        if(!ret.empty())
            ret += delim;
        ret += s;
    }
    return ret;
}

__global__ void parallelKDist(char *data, unsigned int *indices, float*distances, unsigned num_strings, int *suma){
    // each block is comparing a sample with others

    int idx = threadIdx.x+blockDim.x*blockIdx.x;
    if (idx < num_strings && blockIdx.x < sizeof(indices) && threadIdx.x < 64){
        // Fase uno: sumamos todos los valores de la suma de los k-meros de cada entrada.
        // Cada bloque se encarga de calcular la suma de cada entrada
        int entry = blockIdx.x;
        // Cada hilo se encarga de sumar cada permutación.
        // en suma guardamos las apariciones de las 64 posibles combinaciones.
        // Si cada hilo se encargara de cada k-mero no habría por qué hacer la operación atómica.
        const char *currentKmere = c_perms[threadIdx.x];
        // Entonces cada hilo tendría que iterar toda la muestra solo una vez para calcular la suma.
        int entryLength = indices[entry + 1] -  indices[entry];
        // entonces iteramos por cada letra de la entrada hasta la N-k (los índices).
        // Podríamos guardar los índices en memoria constante para agilizar la lectura...
        bool is_same_kmere = true;
        char * sequence = data+indices[entry];
        char currentSubstringFromSample[4];
        for (int i = 0; i < entryLength-3; i++){
            memcpy( currentSubstringFromSample, &sequence[i], 3 );
            currentSubstringFromSample[3] = '\0';
            is_same_kmere = true;
            for(int j = 0; j < 3; j++){
                if (sequence[j] == currentKmere[j]){
                    continue;
                }
                is_same_kmere = false;
                break;
            }
            if(is_same_kmere){
                suma[entry*blockDim.x+threadIdx.x] += 1;
            }
        }
        // Fase dos.
        // Sumamos los mínimos de las cadenas comparadas.
        __syncthreads();
        int nextEntryLength;
        for(int j = entry + 1; j < num_strings - 1; j++){
            nextEntryLength = indices[j + 1] -  indices[j];
            distances[entry*threadIdx.x+j] = 1 - min(suma[entry*blockDim.x+threadIdx.x],suma[j*blockDim.x+threadIdx.x])/ (min(nextEntryLength, entryLength) -3 + 1);

        }
    }


}

int main(int argc, char **argv) {
    //char permutations[len];
    hipError_t error;
    // absolute path of the input data
    string file = "/home/acervantes/kmerDist/plants.fasta";
    //string file = "/home/acervantes/all_seqs.fasta";
    importSeqs(file);

    // Reserving memory for results
    numberOfSequenses = seqs.size();
    //printf("%d sequences founded", numberOfSequenses);
    distancesSequential = (float**) malloc(sizeof(float*) * numberOfSequenses);
    //distancesParallel   = (float**) malloc(sizeof(float*) * numberOfSequenses);
    for(int i = 0; i < numberOfSequenses; i++){
        distancesSequential[i] = (float*) malloc(numberOfSequenses*sizeof(float));
        //distancesParallel[i]   = (float*) malloc(numberOfSequenses*sizeof(float));
    }
    for (int i = 0; i < numberOfSequenses ; i++){
        for (int j = 0; j < numberOfSequenses ; j++) {
            distancesSequential[i][j] = -1;
        }
    }

    doSequentialKmereDistance();

    // Device allocation
    int sumsSize = sizeof(int)*numberOfSequenses*64;
    h_sums = (int*) malloc(sumsSize);
    hipMalloc((void**)&d_sums, sumsSize);

    /* // defining a constant value is passed to the device directly.
    error = hipMemcpyToSymbol(HIP_SYMBOL(c_perms), &perms, 4*64 * sizeof(char) );
    if (error){
        printf("Errorsti : %d: %s\n", error, hipGetErrorString(error));
    }
    */
    error = hipMemcpyToSymbol(HIP_SYMBOL(c_size), &permsSize, sizeof(int) );
    if (error){
        printf("Error %d: %s", error, hipGetErrorString(error));
    }
    unsigned long int sizeDistances = numberOfSequenses*numberOfSequenses * sizeof(float);
    string data_aux = join(seqs, "\0");
    data = data_aux.c_str();
    int indexes[indexes_aux.size()];
    //indexes = (int * ) malloc(indexes_aux.size() * sizeof(int));
    for (int i = 0; i < indexes_aux.size(); i++){
        indexes[i] = indexes_aux[i];
    }
    //std::copy(indexes_aux.begin(), indexes_aux.end(), indexes);

    hipMalloc((void **)&d_data, data_aux.size());
    error = hipMalloc((void **)&d_distances, sizeDistances);
    if (error){
        printf("Error al usar memoria con distancia %d ::", error);
        cout << sizeDistances << endl;
        return 0;
    }
    error = hipMalloc((void **)&d_indices, sizeof(indexes));
    if (error){
        printf("Error malloc %d", error);
    }
    float *h_distances;
    h_distances =(float*) malloc(sizeDistances);

    error = hipMemcpy(d_data, data, data_aux.size(), hipMemcpyHostToDevice);
    if (error){
        printf("Error copying data from host %d", error);
    }
    error = hipMemcpy(d_indices, indexes, sizeof(indexes), hipMemcpyHostToDevice);
    if (error){
        printf("Errorsa %d\n", error);
    }
    int blocks = ceil(seqs.size() / 1024) + 1;
    int threads = 1024;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start,0);
    // Launch kernel
    parallelKDist<<<blocks, threads>>>(d_data, d_indices, d_distances, numberOfSequenses, d_sums);
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    float parallelTimer = 0;
    hipEventElapsedTime(&parallelTimer, start, stop);
    cout<< "Elapsed parallel timer: " << parallelTimer << " ms, " << parallelTimer / 1000 << " secs" <<endl;
    hipMemcpy(h_distances, d_distances, sizeDistances, hipMemcpyDeviceToHost);
    free(distancesSequential);
    //free(distancesParallel);
    hipFree(d_distances);
    hipFree(d_indices);
    hipFree(d_data);
    hipFree(d_sums);
    return 0;
}

void doSequentialKmereDistance(){
    // results files
    FILE *f_seq_res = fopen("/home/acervantes/kmerDist/sequential_results.csv", "w");
    clock_t start_ser = clock();
    sequentialKmerCount(seqs, permutationsList, 3);
    clock_t end_ser = clock();
    double serialTimer = 0;
    serialTimer = double (end_ser-start_ser) / double(CLOCKS_PER_SEC);
    cout << "Elapsed time serial: " << serialTimer << "[s]" << endl;
    for (int i = 0; i < numberOfSequenses ; i++){
        for (int j = 0; j < numberOfSequenses ; j++) {
            fprintf(f_seq_res,"%f ",distancesSequential[i][j]);
            //printf("%f ",distancesSequential[i][j]);
            //distancesParallel[i][j] = 0;
        }
        fprintf(f_seq_res,"\n");
        //printf("\n");
    }
    fclose(f_seq_res);
}

void importSeqs(string inputFile){
    int indexCounter = 0;
    ifstream input(inputFile);
    if (!input.good()) {
        std::cerr << "Error opening: " << inputFile << " . Check your file or pathh." << std::endl;
        exit(0);
    }

    string line;

    bool newSeq = false;
    // Iterate over all secuences
    while (getline(input, line)) {

        // line may be empty so you *must* ignore blank lines
        // or you have a crash waiting to happen with line[0]
        if(line.empty()){
            continue;
        }

        //read the header of
        if (line[0] == '>') {
            // store id
            ids.push_back(line);
            newSeq = true;
        }
        else {
            if (newSeq) {
                seqs.push_back(line);
                indexes_aux.push_back(indexCounter);
                indexCounter += line.size();
                newSeq = false;
            }
            else
                line += line;
            // store seqs

        }
    }
}

void sequentialKmerCount(vector<string> &seqs, vector<string> &permutations , int k){
    string mers[4] = {"A","C","G","T"};
    int numberOfSequences = seqs.size();
    // |kmers| is at most 4**k = 4**3 = 64
    int max_combinations = pow(4,k);
    // Comparing example Ri with R(i+1) until Rn
    for(int i =  0; i < numberOfSequences - 1; i++){
        for(int j = i + 1; j < numberOfSequences; j++){
            if(i >= j)
                continue;
            // iterating over permutations (distance of Ri an Rj).
            int minLength = min(seqs[i].size(), seqs[j].size());
            int sum = 0;
            float distance = -1.0f;
            for(int p = 0; p < max_combinations; p++){
                int minimum = min(
                        permutationsCount(permutations[p], seqs[i],k),
                        permutationsCount(permutations[p], seqs[j],k)
                );
                sum += minimum;
            }

            distance = 1 - (float) sum / (minLength - k + 1);
            distancesSequential[i][j] = distance;
            distancesSequential[j][i] = distance;
        }
    }
    return;
}

int permutationsCount(string permutation, string sequence, int k){
    int sequence_len = sequence.size();
    int counter = 0;
    string current_kmere;
    for(int i = 0; i < sequence_len - k; i++){
        current_kmere = sequence.substr(i,i+k);
        if (permutation.compare(current_kmere) == 0){
            counter++;
        }
    }
    return counter;
}

void getPermutations(char *str, char* permutations, int last, int index){
    string stri;

    int i, len = strlen(str);
    for ( i = 0; i < len; i++ ) {
        permutations[index] = str[i] ;
        if (index == last){
            stri = permutations;
            permutationsList.push_back(stri);
        }
        else
            getPermutations (str, permutations, last, index+1);
    }
}

void printSeqs(){
    cout<< "total number of seqs: " << seqs.size() << endl;
    for (int i = 0; i<seqs.size(); i++){
        cout << ">" <<  seqs[i] << endl;
    }
}

/*
void kmerDistance(int k){

}
 */