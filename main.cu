#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <set>
#include <math.h>
#include <algorithm>
#include <sstream>
#include <typeinfo>
#include "hip/hip_runtime.h"
#include <hip/hip_cooperative_groups.h>
#include <limits.h>

using namespace std;
#define PERMS_KMERES 64
int          numberOfSequenses = 0;
char *       all_seqs;
unsigned int size_all_seqs = 0;


// Method definition
void importSeqs(string inputFile);
void printSeqs();
void getPermutations(char *str, char* permutations, int last, int index);
int permutationsCount(string permutation, string sequence, int k);
void sequentialKmerCount(vector<string> &seqs, vector<string> &permutations , int k);
void doParallelKmereDistance();
void doSequentialKmereDistance();
// Vectors to store ids and seqs
vector<string> ids;
vector<string> seqs;
vector<int> indexes_aux;

// Device variables.
char    * d_data; // all the strings.
int     * d_indices;
float   * d_distances;
int * d_sums; // coincidences of k-mer on each input
int * d_mins, *h_mins;
int * h_sums;
int minsSize;

string file = "/home/acervantes/kmerDist/plants.fasta";
//string file = "/home/acervantes/kmerDist/all_seqs.fasta";

// number of permutations of RNA K_meres and k-value
__constant__ char c_perms[64][4] = {
        "AAA", "AAC", "AAG","AAT",
        "ACA", "ACC", "ACG","ACT",
        "AGA", "AGC", "AGG", "AGT",
        "ATA", "ATC", "ATG", "ATT",

        "CAA", "CAC", "CAG", "CAT",
        "CCA", "CCC", "CCG", "CCT",
        "CGA", "CGC", "CGG", "CGT",
        "CTA", "CTC", "CTG", "CTT",

        "GAA", "GAC", "GAG", "GAT",
        "GCA", "GCC", "GCG", "GCT",
        "GGA", "GGC", "GGG", "GGT",
        "GTA", "GTC", "GTG", "GTT",

        "TAA", "TAC", "TAG", "TAT",
        "TCA", "TCC", "TCG", "TCT",
        "TGA", "TGC", "TGG", "TGT",
        "TTA", "TTC", "TTG", "TTT",
};

char perms[64][4] = {
        "AAA", "AAC", "AAG","AAT",
        "ACA", "ACC", "ACG","ACT",
        "AGA", "AGC", "AGG", "AGT",
        "ATA", "ATC", "ATG", "ATT",

        "CAA", "CAC", "CAG", "CAT",
        "CCA", "CCC", "CCG", "CCT",
        "CGA", "CGC", "CGG", "CGT",
        "CTA", "CTC", "CTG", "CTT",

        "GAA", "GAC", "GAG", "GAT",
        "GCA", "GCC", "GCG", "GCT",
        "GGA", "GGC", "GGG", "GGT",
        "GTA", "GTC", "GTG", "GTT",

        "TAA", "TAC", "TAG", "TAT",
        "TCA", "TCC", "TCG", "TCT",
        "TGA", "TGC", "TGG", "TGT",
        "TTA", "TTC", "TTG", "TTT",
};
int permsSize = sizeof(perms) ;

vector<string> permutationsList (perms, end(perms));

float ** distancesSequential;
float ** distancesParallel;
/*
string join(const vector<string>& vec, const char* delim){
    stringstream res;
    copy(vec.begin(), vec.end(), ostream_iterator<string>(res, delimiter.c_str()));
    return res.str();
}
 */
string join(const std::vector<std::string> &lst, const std::string &delim){
    std::string ret;
    for(const auto &s : lst) {
        if(!ret.empty())
            ret += delim;
        ret += s;
    }
    return ret;
}
/**
 *
 * @param data:    buffer con todas las cadenas
 * @param indices: índices donde inicia cada cadena nueva en los datos
 * @param distances: matriz resultante de las distancias
 * @param num_seqs: número de cadenas de entrada.
 * @param suma: Matriz de rxc donde cada renglón equivale a las coincidencias de cada k-mero
 *              en una cadena de entrada (columna).
 */
__global__ void sumKmereCoincidences(char *data, int *indices, unsigned num_seqs, int *sum){
    // each block is comparing a sample with others
    //int idx = threadIdx.x+blockDim.x*blockIdx.x;
    int entry = blockIdx.x;
    // Each thread count all coincidences of a k-mere combination.
    int k_mere = threadIdx.x;
    //int idx    = blockIdx.x * blockDim.x + threadIdx.x;
    int idx    = blockIdx.x * PERMS_KMERES + threadIdx.x;
    extern __shared__ int sDataSum[];
    int maxIdxSM = (int) (49152 / sizeof(int));
    //printf("outside blockid: %d \n", blockIdx.x);
    // 12288
    if ((blockIdx.x < num_seqs) && (threadIdx.x < PERMS_KMERES)){
        if(idx < maxIdxSM )
            sDataSum[idx] = 0;
        __syncthreads();
        // Fase uno: sumamos todos los valores de la suma de los k-meros de cada entrada.
        // Cada bloque se encarga de cada cadena de entrada
        // Cada hilo se encarga de sumar cada permutación.
        const char *currentKmere = c_perms[k_mere];
        // Entonces cada hilo tendría que iterar toda la muestra solo una vez para calcular la suma.
        int entryLength = indices[entry + 1] -  indices[entry];
        // entonces iteramos por cada letra de la entrada hasta la N-k (los índices).
        // Podríamos guardar los índices en memoria constante para agilizar la lectura...
        bool is_same_kmere = true;
        char * sequence = data+indices[entry];
        char currentSubstringFromSample[4];
        for (int i = 0; i < entryLength-3; i++){
            memcpy( currentSubstringFromSample, &sequence[i], 3 );
            currentSubstringFromSample[3] = '\0';
            is_same_kmere = true;
            for(int j = 0; j < 3; j++){
                if (currentSubstringFromSample[j] == currentKmere[j]){
                    continue;
                }
                is_same_kmere = false;
                break;
            }
            if(is_same_kmere){
                if(idx < maxIdxSM)
                    sDataSum[idx] += 1;
                else
                    sum[idx] += 1;
            }
        }
        __syncthreads();
        if (idx < maxIdxSM)
            sum[idx] = sDataSum[idx];
    }
}
// intento 1: .350 [s]
__global__ void sumKmereCoincidencesGlobalMemory(char *data, int *indices, unsigned num_seqs, int *sum){
    // each block is comparing a sample with others
    //int idx = threadIdx.x+blockDim.x*blockIdx.x;
    int entry = blockIdx.x;
    // Each thread count all coincidences of a k-mere combination.
    int k_mere = threadIdx.x;
    //int idx    = blockIdx.x * blockDim.x + threadIdx.x;
    int idx    = blockIdx.x * PERMS_KMERES + threadIdx.x;
    //printf("outside blockid: %d \n", blockIdx.x);
    if ((entry < num_seqs) && (threadIdx.x < PERMS_KMERES)){

        // Fase uno: sumamos todos los valores de la suma de los k-meros de cada entrada.
        // Cada bloque se encarga de cada cadena de entrada
        // Cada hilo se encarga de sumar cada permutación.
        const char *currentKmere = c_perms[k_mere];
        // Entonces cada hilo tendría que iterar toda la muestra solo una vez para calcular la suma.
        int entryLength = indices[entry + 1] -  indices[entry];
        // entonces iteramos por cada letra de la entrada hasta la N-k (los índices).
        // Podríamos guardar los índices en memoria constante para agilizar la lectura...
        bool is_same_kmere = true;
        char * sequence = data+indices[entry];
        char currentSubstringFromSample[4];
        for (int i = 0; i < entryLength-3; i++){
            memcpy( currentSubstringFromSample, &sequence[i], 3 );
            currentSubstringFromSample[3] = '\0';
            is_same_kmere = true;
            for(int j = 0; j < 3; j++){
                if (currentSubstringFromSample[j] == currentKmere[j]){
                    continue;
                }
                is_same_kmere = false;
                break;
            }
            if(is_same_kmere){
                sum[idx] += 1;
            }
        }
        __syncthreads();
    }
}


/*
__global__ void minKmerzeDist(int *sums, double *distances, int num_seqs, double *mins){
    // Cada bloque se encargará de calcular un k-mero
    // Los hilos distribuirán la tarea de calcular el mínimo entre las entradas.
    // Distances poddría ser una matriz triangular o podría ser un arreglo cuyo acceso sea por una función
    // hash para ahorrar memoria.

    // En el bloque comparamos todas las coincidencias con los k-meros
    int current_kmere = blockIdx.x;
    int current_seq   = threadIdx.x;
    int idx           = blockIdx.x*blockDim.x+threadIdx.x;
    int comparisons   = current_seq - num_seqs;
    int min           = INT_MAX;
    // TODO: caso para más de 1024 entradas, se necesita seguir ejecutando hasta que todas las entradas se puedan comparar
    if(current_seq < num_seqs - 1){
        // Cada iteración es la comparación desde la secuencia 'start' hasta la última.
        for(int start = 0; start < num_seqs - 1; start++){
            //Se calculan los mínimos de una entrada contra las restantes
             if (start < current_seq)
                min = (sums[idx+start] < sums[idx + start + current_seq + 1]) ? sums[idx+start] : sums[idx + start + current_seq + 1];
            __syncthreads();

        }
    }
    __syncthreads();

    mins[idx] =

}*/

// https://www.geeksforgeeks.org/convert-given-upper-triangular-matrix-to-1d-array/
__device__ __host__ int getIdxTriangularMatrixRowMajor(int i, int j, int n){
    return (n * (i - 1) - (((i - 2) * (i - 1)) / 2)) + (j - i);
}

// extracted and modified from MK Programming Massively. 2nd Edition p.209
__global__ void parallelSum(float *results, int idxResult, int InputSize) {
    __shared__ int XY[PERMS_KMERES];
    extern __shared__ int min_sums[];
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i < InputSize) {
        XY[threadIdx.x] = min_sums[i];
    }
    for (unsigned int stride = 1; stride < blockDim.x; stride *= 2) {
        __syncthreads();
        int index = (threadIdx.x+1) * 2* stride -1;
        if (index < blockDim.x) {
            XY[index] += XY[index - stride];
        }
    }
    for (int stride = PERMS_KMERES/4; stride > 0; stride /= 2) {
        __syncthreads();
        int index = (threadIdx.x+1)*stride*2 - 1;
        if(index + stride < PERMS_KMERES) {
            XY[index + stride] += XY[index];
        }
    }
    __syncthreads();
    if (i == 0)
        results[idxResult] = XY[InputSize-1];
}

__global__ void minKmereDist(int *sums, float *distances, int num_seqs, int start){
    int current_seq   = blockIdx.x;
    int idx           = current_seq+blockDim.x*threadIdx.x;
    //int idxDist       = start*blockDim.x+current_seq;
    int min_sums[PERMS_KMERES] = {0};
    // Se guarda en memoria compartida las repeticiones de los kmeros de las dos entradas a comparar.
    __shared__ int seqs_sums[PERMS_KMERES*2];
    //TODO: ¿la variable totalsum pueden verlos los demás bloques?
    //int totalSum;

    if(current_seq < num_seqs - 1){

        for(int i = 0, j=0; i < PERMS_KMERES; i++){
            seqs_sums[j++]   = sums[idx];
            seqs_sums[j++]   = sums[idx+1];
        }
        __syncthreads();
        for(int i = 0; i < PERMS_KMERES; i++){
            if (seqs_sums[2*i] < seqs_sums[2*i+1]){
                min_sums[i] += seqs_sums[2*i];
            }
            else{
                printf("");
                min_sums[i] += seqs_sums[2*i+1];
            }
        }
        __syncthreads();
        //parallelSum<<<1,64>>>(distances,idxDist+1, PERMS_KMERES);
    }

}
/*
 * @param sums:    matriz que contiene el número de coincidencias de cada k-mero en cada cadena.
 * @param mins:    matriz de las sumas de los k-meros mínimos entre las cadenas i e i+1, respectivamente
 *                  (tamaño num_seqsxnum_seqs).
 *                  Por el momento se implementó un arreglo equivalente a una matriz triangular con el fin
 *                   de reducir la memoria.
 * @param num_seqs: secuencias totales a considerar.
 * @param num_kmeres: combinaciones totales de k-meros.
 *
 */
//  TODO: checar si num_kmeres se puede usar para inicializar un arreglo aquí.
__global__ void minKmeres(int *sums, int *mins, int num_seqs, int num_kmeres){
    int current_seq   = blockIdx.x;
    int current_kmere  = threadIdx.x;
    int current_comp;
    int idxMin;
    // es del tamaño de la ent
    __shared__ int PIVOT[PERMS_KMERES];
    int i;
    //j;
    int jump = 1;
    int current_sum = 0;

    // Guardamos en memoria compartida el pivote porque se accederá dos veces.
    if(current_seq < num_seqs && current_kmere < num_kmeres) {// initialize PIVOT
        PIVOT[current_kmere] = sums[current_seq+current_kmere*blockDim.x];
        __syncthreads();
        while(jump+current_seq < num_seqs){
            for(i = current_seq; i < num_seqs; i+=jump){
                current_comp = sums[current_seq+i+current_kmere*num_seqs];
                current_sum  = current_comp < PIVOT[current_kmere] ?
                               current_comp : PIVOT[current_kmere];
            }
            jump++;
        }
        __syncthreads(); // TODO: verificar que no afecta esto al resultado, pero reduce las colisiones.
        idxMin = getIdxTriangularMatrixRowMajor(current_seq+1, current_comp+1, num_kmeres);
        atomicAdd(&mins[idxMin], current_sum);
    }
}



int main(int argc, char **argv) {
    //char permutations[len];
    // absolute path of the input data
    importSeqs(file);
    std::cout << "Size all seqs:" << size_all_seqs << std::endl;
    // Device allocation
    doParallelKmereDistance();
    return 0;
}

void doSequentialKmereDistance(){
    // results files
    FILE *f_seq_res = fopen("/home/acervantes/kmerDist/sequential_results.csv", "w");

    distancesSequential = (float**) malloc(sizeof(float*) * numberOfSequenses);
    //distancesParallel   = (float**) malloc(sizeof(float*) * numberOfSequenses);
    for(int i = 0; i < numberOfSequenses; i++){
        distancesSequential[i] = (float*) malloc(numberOfSequenses*sizeof(float));
        //distancesParallel[i]   = (float*) malloc(numberOfSequenses*sizeof(float));
    }
    for (int i = 0; i < numberOfSequenses ; i++){
        for (int j = 0; j < numberOfSequenses ; j++) {
            distancesSequential[i][j] = -1;
        }
    }
    clock_t start_ser = clock();
    sequentialKmerCount(seqs, permutationsList, 3);
    clock_t end_ser = clock();
    double serialTimer = 0;
    serialTimer = double (end_ser-start_ser) / double(CLOCKS_PER_SEC);
    cout << "Elapsed time serial: " << serialTimer << "[s]" << endl;

    for (int i = 0; i < numberOfSequenses ; i++){
        for (int j = 0; j < numberOfSequenses ; j++) {
            fprintf(f_seq_res,"%f ",distancesSequential[i][j]);
            //printf("%f ",distancesSequential[i][j]);
            //distancesParallel[i][j] = 0;
        }
        fprintf(f_seq_res,"\n");
        //printf("\n");
    }
    fclose(f_seq_res);
}

void doParallelKmereDistance(){
    hipError_t error;
    /**
     * Inicialización
     * */

    // Buffer de cadenas
    error = hipMalloc((void **)&d_data, size_all_seqs);
    if (error){
        printf("Error #%d allocating device memory with data.", error);
        exit(1);
    }
    error = hipMemcpy(d_data, all_seqs, size_all_seqs, hipMemcpyHostToDevice);
    if (error){
        printf("Error copying data from host %d\n", error);
        exit(1);
    }

    // Los índices de las entradas de las cadenas.
    int *indexes = (int *) malloc((int)indexes_aux.size() * sizeof(int));
    for (int i = 0; i < indexes_aux.size(); i++){
        indexes[i] = indexes_aux[i];
    }
    int indexesBytesSize = indexes_aux.size()*sizeof(int);
    error = hipMalloc((void **)&d_indices, indexesBytesSize);
    if (error){
        printf("Error malloc %d", error);
    }
    error = hipMemcpy(d_indices, indexes, indexesBytesSize, hipMemcpyHostToDevice);
    if (error) printf("Error copying data from device %d\n", error);

    // Suma de cada kmero de cada entrada.
    int numSumResults = numberOfSequenses*PERMS_KMERES; // sequences x 4**3
    int sumsSize = sizeof(int)*numSumResults;
    h_sums = (int*) malloc(sumsSize);
    for (int i = 0; i < numSumResults; i++){
        h_sums[i] = 0;
    }
    hipMalloc((void **)&d_sums, sumsSize);
    error = hipMemcpy(d_sums, h_sums, sumsSize, hipMemcpyHostToDevice);
    if (error){
        printf("Error copying data from host %d", error);
        exit(1);
    }
    /* // defining a constant value is passed to the device directly.
    error = hipMemcpyToSymbol(HIP_SYMBOL(c_perms), &perms, 4*64 * sizeof(char) );
    if (error){
        printf("Errorsti : %d: %s\n", error, hipGetErrorString(error));
    }
    */
    //int indexes[indexes_aux.size()];

    /**
     * Mins: será una estructura de datos para minimizar el uso de memoria.
     * En total, sería de n (n+1) / 2 donde n es el número de muestras.
     * */
    minsSize = (int) (numberOfSequenses*(numberOfSequenses+1) / 2);
    h_mins   = (int*) malloc(minsSize * sizeof(int));
    for(int i = 0; i < minsSize; i++)
        h_mins[i] = 0;
    int threads = 64;
    int blocks = numberOfSequenses;
    //int blocks = 10;
    //int threads = 64;
    hipEvent_t start;
    hipEvent_t stop;
    hipError_t err_;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    printf("Running %d blocks and %d threads\n", blocks, threads);
    // Launch kernel
    //int smSize = 49152;
    //sumKmereCoincidences<<<blocks, threads, smSize>>>(d_data, d_indices, numberOfSequenses, d_sums);
    hipEventRecord(start,0);
    sumKmereCoincidencesGlobalMemory<<<blocks, threads>>>(d_data, d_indices, numberOfSequenses, d_sums);
    //minKmereDist<<<10, 1024>>>(d_sums,d_distances, numberOfSequenses, 0);
    hipDeviceSynchronize();
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    float parallelTimer = 0;
    hipEventElapsedTime(&parallelTimer, start, stop);
    cout<< "Elapsed parallel timer: " << parallelTimer << " ms, " << parallelTimer / 1000 << " secs" <<endl;
    err_ = hipGetLastError();
    if (err_)
        printf("LastError kmere dist: %d\n", err_);
    //minKmeres<<<blocks, threads>>>(d_sums, d_mins, numberOfSequenses, PERMS_KMERES);
    //hipDeviceSynchronize();
    err_ = hipGetLastError();
    if (err_)
        printf("LastError min kmeres: %d\n", err_);

    //unsigned long int sizeDistances = numberOfSequenses*numberOfSequenses * sizeof(float);
    //hipMemcpy(h_distances, d_distances, sizeDistances, hipMemcpyDeviceToHost);

    hipMemcpy(h_sums, d_sums, sumsSize, hipMemcpyDeviceToHost);
    /*
    printf("Sums:\n");
    for(int j = 0, idx = 0; j < 64; j++){
        printf("%d: ", j);
        for(int i = 0; i < numberOfSequenses; i++){
            printf("%d,\t", h_sums[idx++]);
        }
        printf("\n");
    }*/

    free(distancesSequential);
    //free(distancesParallel);
    hipFree(d_distances);
    hipFree(d_indices);
    hipFree(d_data);
    hipFree(d_sums);
    hipFree(d_mins);

    return;
    /*
    error = hipMalloc((void **)&d_distances, sizeDistances);
    if (error){
        printf("Error al usar memoria con distancia %d ::", error);
        cout << sizeDistances << endl;
        return 0;
    }*/


    //    float *h_distances;
    //    h_distances =(float*) malloc(sizeDistances);
    //    int dimsDistances = numberOfSequenses*numberOfSequenses;
    //    for(int i=0; i<dimsDistances; i++){
    //        h_distances[i] = 0;
    //    }


    /*
    error = hipMemcpy(d_distances, h_distances, sizeDistances, hipMemcpyHostToDevice);
    if (error){
        printf("Error copying distances matrix from host %d", error);
    }*/

    /*
    error = hipMalloc((void **)&d_mins, minsSize*sizeof(int));
    if (error){
        printf("Error #%d allocating memory to d_mins", error);
        exit(1);
    }
    error = hipMemcpy(d_mins, h_mins, minsSize*sizeof(int), hipMemcpyHostToDevice);
     if (error){
        printf("Error copying data from host %d", error);
        exit(1);
    }
     */

}
void importSeqs(string inputFile){
    int indexCounter = 0;
    ifstream input(inputFile);
    if (!input.good()) {
        std::cerr << "Error opening: " << inputFile << " . Check your file or pathh." << std::endl;
        exit(0);
    }
    string line;
    string acc = "";
    string globalAcc = "";
    bool newSeq = false;

    // Iterate over all secuences
    while (getline(input, line)) {

        // line may be empty so you *must* ignore blank lines
        // or you have a crash waiting to happen with line[0]
        if(line.empty()){
            continue;
        }
        //read the header of
        if (line[0] == '>') {
            // store id
            ids.push_back(line);
            newSeq = true;
            continue;
        }
        if (newSeq) {
            newSeq = false;
            acc = line;
            while (getline(input, line)) {
                if(line.empty() || line[0] == 13){
                    acc += "|";
                    seqs.push_back(acc);
                    indexes_aux.push_back(indexCounter);
                    indexCounter += acc.size();
                    globalAcc += acc;
                    acc = "";
                    break;
                }
                acc += line;
            }
            if (acc != ""){
                acc += "|";
                seqs.push_back(acc);
                indexes_aux.push_back(indexCounter);
                indexCounter += acc.size();
                globalAcc += acc;
                acc = "";
                indexes_aux.push_back(indexCounter);
            }
        }
    }
    int last_index = indexes_aux.size();
    numberOfSequenses = seqs.size();
    size_all_seqs = globalAcc.size()*sizeof(char);
    all_seqs = (char *) malloc(size_all_seqs);
    for(int i = 0; i < globalAcc.size(); i++){
        if (globalAcc[i] == '|'){
            all_seqs[i] = '\0';
            continue;
        }
        all_seqs[i] = globalAcc[i];
    }
    return;
}

void sequentialKmerCount(vector<string> &seqs, vector<string> &permutations , int k){
    string mers[4] = {"A","C","G","T"};
    int numberOfSequences = seqs.size();
    // |kmers| is at most 4**k = 4**3 = 64
    int max_combinations = pow(4,k);
    // Comparing example Ri with R(i+1) until Rn
    for(int i =  0; i < numberOfSequences - 1; i++){
        for(int j = i + 1; j < numberOfSequences; j++){
            if(i >= j)
                continue;
            // iterating over permutations (distance of Ri an Rj).
            int minLength = min(seqs[i].size(), seqs[j].size());
            int sum = 0;
            float distance = -1.0f;
            for(int p = 0; p < max_combinations; p++){
                int minimum = min(
                        permutationsCount(permutations[p], seqs[i],k),
                        permutationsCount(permutations[p], seqs[j],k)
                );
                sum += minimum;
            }

            distance = 1 - (float) sum / (minLength - k + 1);
            distancesSequential[i][j] = distance;
            distancesSequential[j][i] = distance;
        }
    }
    return;
}

int permutationsCount(string permutation, string sequence, int k){
    int sequence_len = sequence.size();
    int counter = 0;
    string current_kmere;
    for(int i = 0; i < sequence_len - k; i++){
        current_kmere = sequence.substr(i,i+k);
        if (permutation.compare(current_kmere) == 0){
            counter++;
        }
    }
    return counter;
}

void getPermutations(char *str, char* permutations, int last, int index){
    string stri;

    int i, len = strlen(str);
    for ( i = 0; i < len; i++ ) {
        permutations[index] = str[i] ;
        if (index == last){
            stri = permutations;
            permutationsList.push_back(stri);
        }
        else
            getPermutations (str, permutations, last, index+1);
    }
}

void printSeqs(){
    cout<< "total number of seqs: " << seqs.size() << endl;
    for (int i = 0; i<seqs.size(); i++){
        cout << ">" <<  seqs[i] << endl;
    }
}
