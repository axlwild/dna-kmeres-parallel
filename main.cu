#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <set>
#include <math.h>
#include <algorithm>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <sstream>

using namespace std;
// Method definition
void importSeqs(string inputFile);
void printSeqs();
void getPermutations(char *str, char* permutations, int last, int index);
int permutationsCount(string permutation, string sequence, int k);
void sequentialKmerCount(vector<string> &seqs, vector<string> &permutations , int k);
// Vectors to store ids and seqs
vector<string> ids;
vector<string> seqs;
vector<int> indexes_aux;
int size;
int permutationsSize;
int countArraySize;


// Device variables.
char * d_data; // all the strings.
unsigned int * d_indices;
float * d_distances;
const char * data;

__constant__ const char* c_perms[] = {"AAA", "AAC", "AAT","AAG",
                                      "ACA", "ACC", "ACT","ACG",
                                      "ATA", "ATC", "ATT","ATG",
                                      "AGA", "AGC", "AGT","AGG",

                                      "CAA", "CAC", "CAT","CAG",
                                      "CCA", "CCC", "CCT","CCG",
                                      "CTA", "CTC", "CTT","CTG",
                                      "CGA", "CGC", "CGT","CGG",

                                      "GAA", "GAC", "GAT","GAG",
                                      "GCA", "GCC", "GCT","GCG",
                                      "GTA", "GTC", "GTT","GTG",
                                      "GGA", "GGC", "GGT","GGG",

                                      "TAA", "TAC", "TAT","TAG",
                                      "TCA", "TCC", "TCT","TCG",
                                      "TTA", "TTC", "TTT","TTG",
                                      "TGA", "TGC", "TGT","TGG",
};
__constant__ int  c_size ;
const char * perms[] = {"AAA", "AAC", "AAT","AAG",
                      "ACA", "ACC", "ACT","ACG",
                      "ATA", "ATC", "ATT","ATG",
                      "AGA", "AGC", "AGT","AGG",

                      "CAA", "CAC", "CAT","CAG",
                      "CCA", "CCC", "CCT","CCG",
                      "CTA", "CTC", "CTT","CTG",
                      "CGA", "CGC", "CGT","CGG",

                      "GAA", "GAC", "GAT","GAG",
                      "GCA", "GCC", "GCT","GCG",
                      "GTA", "GTC", "GTT","GTG",
                      "GGA", "GGC", "GGT","GGG",

                      "TAA", "TAC", "TAT","TAG",
                      "TCA", "TCC", "TCT","TCG",
                      "TTA", "TTC", "TTT","TTG",
                      "TGA", "TGC", "TGT","TGG",
};
int permsSize = sizeof(perms) / sizeof("AAA");

vector<string> permutationsList (perms, end(perms));

float ** distancesSequential;
float ** distancesParallel;

string join(const vector<string>& vec, const char* delim){
    stringstream res;
    copy(vec.begin(), vec.end(), ostream_iterator<string>(res, delim));
    return res.str();
}

__global__ void parallelKDist(char *data, unsigned int *indices, float*distances, unsigned num_strings){
    int idx = threadIdx.x+blockDim.x*blockIdx.x;
    char * sequence = data+indices[idx] ;
    bool is_coincidence = true;
    if (idx < num_strings && blockIdx.x < sizeof(indices)){
        // iterating over permutations
        printf("soy  el índice %d\n y c_size vale %d\n", idx, c_size);
        printf("Sequence vale: %s", sequence);
        for(int i = 0 ; i < c_size; i++){
            for(int j = 0; j < 3; j++){

                //c_perms[i][j]
                printf("Secuencia: %c\n",sequence[j]);

                if (sequence[j] != c_perms[i][j]){
                    is_coincidence = false;
                    continue;
                }
            }
            if(is_coincidence){
                atomicAdd(&(distances[idx]), 1);
            }
        }
    }
}

int main(int argc, char **argv) {
    //char permutations[len];
    int error;
    int numberOfSequenses = 0;
    //getPermutations(chars, permutations, len - 1, 0);
    /*
     * int len = strlen("ACGT") ;
     * for (int i = 0; i < permutationsList.size() ; i++){
        cout << permutationsList.at(i) << endl;
    }
    */
    // absolute path of the input data
    string file = "/home/acervantes/plants.fasta";
    //string file = "/home/acervantes/all_seqs.fasta";
    importSeqs(file);
    // Reserving memory for resultsf
    numberOfSequenses = seqs.size();
    distancesSequential = (float**) malloc(sizeof(float*) * numberOfSequenses);
    //distancesParallel   = (float**) malloc(sizeof(float*) * numberOfSequenses);
    for(int i = 0; i < numberOfSequenses; i++){
        distancesSequential[i] = (float*) malloc(numberOfSequenses*sizeof(float));
        //distancesParallel[i]   = (float*) malloc(numberOfSequenses*sizeof(float));
    }
    for (int i = 0; i < numberOfSequenses ; i++){
        for (int j = 0; j < numberOfSequenses ; j++) {
            distancesSequential[i][j] = 0;
            //distancesParallel[i][j] = 0;
        }
    }
    /*
    sequentialKmerCount(seqs, permutationsList, 3);
    for (int i = 0; i < numberOfSequenses ; i++){
        for (int j = 0; j < numberOfSequenses ; j++) {
            cout << distancesSequential[i][j] << "\t";
        }
        cout << endl;
    }
     */

    // Device allocation
    /*
    std::ostringstream data_aux;
    const char * data = data_aux.c_str()
    std::copy(seqs.begin(), seqs.end(), std::ostream_iterator<std::string>(imploded, "\0"));
     */
    error = hipMemcpyToSymbol(HIP_SYMBOL(c_perms), perms, sizeof(perms) / sizeof("AAA") );
    if (error){
        printf("Errors %d", error);
    }
    error = hipMemcpyToSymbol(HIP_SYMBOL(c_size), &permsSize, sizeof(int) );
    if (error){
        printf("Errors %d", error);
    }

    int sizeDistances = numberOfSequenses*numberOfSequenses * sizeof(float);
    string data_aux = join(seqs, "\0");
    data = data_aux.c_str();
    int indexes[indexes_aux.size()];
    //indexes = (int * ) malloc(indexes_aux.size() * sizeof(int));
    for (int i = 0; i < indexes_aux.size(); i++){
        indexes[i] = indexes_aux[i];
    }
    //std::copy(indexes_aux.begin(), indexes_aux.end(), indexes);

    hipMalloc((void **)&d_data, data_aux.size());
    hipMalloc((void **)&d_distances, sizeDistances);
    error = hipMalloc((void **)&d_indices, sizeof(indexes));
    if (error){
        printf("Errors %d", error);
    }
    float *h_distances;
    h_distances =(float*) malloc(sizeDistances);

    error = hipMemcpy(d_data, data, sizeof(data), hipMemcpyHostToDevice);
    if (error){
        printf("Error %d", error);
    }
    error = hipMemcpy(d_indices, indexes, sizeof(indexes), hipMemcpyHostToDevice);
    if (error){
        printf("Errorsa %d\n", error);
    }
    int blocks = ceil(seqs.size() / 1024) + 1;
    int threads = 1024;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start,0);
    // Launch kernel
    parallelKDist<<<blocks, threads>>>(d_data, d_indices, d_distances, numberOfSequenses);
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    float parallelTimer = 0;
    hipEventElapsedTime(&parallelTimer, start, stop);
    cout<< "Elapsed parallel timer: " << parallelTimer << " ms, " << parallelTimer / 1000 << " secs" <<endl;
    hipMemcpy(h_distances, d_distances, sizeDistances, hipMemcpyDeviceToHost);
    free(distancesSequential);
    //free(distancesParallel);
    hipFree(d_distances);
    hipFree(d_indices);
    hipFree(d_data);
    return 0;
}

void importSeqs(string inputFile){
    int indexCounter = 0;
    ifstream input(inputFile);
    if (!input.good()) {
        std::cerr << "Error opening: " << inputFile << " . Check your file or pathh." << std::endl;
        exit(0);
    }

    string line;

    bool newSeq = false;
    // Iterate over all secuences
    while (getline(input, line)) {

        // line may be empty so you *must* ignore blank lines
        // or you have a crash waiting to happen with line[0]
        if(line.empty()){
            continue;
        }

        //read the header of
        if (line[0] == '>') {
            // store id
            ids.push_back(line);
            newSeq = true;
        }
        else {
            if (newSeq) {
                seqs.push_back(line);
                indexes_aux.push_back(indexCounter);
                indexCounter += line.size();
                newSeq = false;
            }
            else
                line += line;
            // store seqs

        }
    }
}

void sequentialKmerCount(vector<string> &seqs, vector<string> &permutations , int k){
    string mers[4] = {"A","C","G","T"};
    int numberOfSequences = seqs.size();
    // |kmers| is at most 4**k = 4**3 = 64
    int max_combinations = pow(4,k);
    // Comparing example Ri with i+1 until Rn
    for(int i =  0; i < numberOfSequences - 1; i++){
        for(int j = i + 1; j < numberOfSequences; j++){
            if(i == j)
                continue;
            // iterating over permutations (distance of Ri an Rj).
            int minLength = min(seqs[i].size(), seqs[j].size());
            int sum = 0;
            float distance = -1.0f;
            for(int p = 0; p < max_combinations; p++){
                int minimum = min(
                    permutationsCount(permutationsList[p], seqs[i],k),
                    permutationsCount(permutationsList[p], seqs[j],k)
                );
                sum += minimum;
            }

            distance = 1 - (float) sum / (minLength - k + 1);
            distancesSequential[i][j] = distance;
            distancesSequential[j][i] = distance;
        }
    }
    return;
}

int permutationsCount(string permutation, string sequence, int k){
    int sequence_len = sequence.size();
    int counter = 0;
    string current_kmere;
    for(int i = 0; i < sequence_len - k; i++){
        current_kmere = sequence.substr(i,i+k);
        if (permutation.compare(current_kmere) == 0){
            counter++;
        }
    }
    return counter;
}

void getPermutations(char *str, char* permutations, int last, int index){
    string stri;

    int i, len = strlen(str);
    for ( i = 0; i < len; i++ ) {
        permutations[index] = str[i] ;
        if (index == last){
            stri = permutations;
            permutationsList.push_back(stri);
        }
        else
            getPermutations (str, permutations, last, index+1);
    }
}

void printSeqs(){
    cout<< "total number of seqs: " << seqs.size() << endl;
    for (int i = 0; i<seqs.size(); i++){
        cout << ">" <<  seqs[i] << endl;
    }
}

/*
void kmerDistance(int k){

}
 */