#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <set>
#include <algorithm>
#include <sstream>
#include <typeinfo>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "hip/hip_runtime.h"
#include <hip/hip_cooperative_groups.h>
#include <climits>
#include "utils.h"
#include <map>
#include "kernels.h"
#include "utils.cpp"

#ifndef PERMS_KMERES
#define PERMS_KMERES 64
#endif


#define THREADS 64
#define N (54018*1024*128)
#define PRINT_ANSWERS false
#define PRINT_ANSWERS_FILE true

#define BLOCKS_STEP_1 40000
#define MAX_SEQS 1000
#define VERBOSE true

using namespace std;
int          numberOfSequenses = 0;
unsigned int size_all_seqs = 0;

long threads = THREADS;
//long blocks = 32768; // 2.61
//long blocks = 30000; // 2.32
//long blocks = 5000; // 2.0511
long blocks = 1000; // 2.012

// TODO: dinámico, si el número de hilos necesario es menor de 1024, usar esa cantidad
int threadsStep1 = 1024;
int blockThread1 = BLOCKS_STEP_1;
bool bug_log = false;
string file = "/home/acervantes/kmerDist/plants.fasta";
//string file = "/home/acervantes/kmerDist/all_seqs.fasta";
// to run this, execute importSeqsNoNL.
//string file = "/home/acervantes/kmerDist/genomic.fna";
// Method definition
void importSeqs(string inputFile); 
void importSeqsNoNL(string inputFile);

void printSeqs();
void getPermutations(char *str, char* permutations, int last, int index);


void sequentialKmerCount2(vector<string> &seqs, vector<string> &permutations , int k);
void doParallelKmereDistance();
void doSequentialKmereDistance();
void permutationsCountAll(string sequence, int * countResults, int max_combinations, int k);
long getIdxTriangularMatrixRowMajorSeq(long i, long j, long n);
// Vectors to store ids and seqs
vector<string> ids;
vector<string> seqs;
vector<int> indexes_aux;
// Device variables.
char    *data; // all the strings.
int     *indexes;
float   *distances;
int     *sums; // coincidences of k-mer on each input
float   *mins;
long minsSize;
long resultsArraySize;


// 4 cadenas sería 1 bit
// 1111 1111 = 1 byte
// 00 -> A
// 01 -> C
// 10 -> G
// 11 -> T

// AACG -> 00000110
// AACGA -> 00000110 00__ ____

//__constant__ char c_perms[][4];
//char perms[PERMS_KMERES][K+1];



std::map<std::string, int> permutationsMap;
vector<string> permutationsList;
char **perms;
float * distancesSequential;

int main() {

    const char *alphabet = "ACGT";
    int sizeAlphabet = 4;
    int permsSize    = pow(sizeAlphabet, K);
    perms = (char**) malloc(permsSize * sizeof(char*));
    for(int i = 0; i < permsSize; i++)
    {
        perms[i] = (char*) malloc((K+1)*sizeof(char));
    }
    permutation(alphabet, K, perms);
    for(int i = 0; i < permsSize; i++){
        permutationsList.push_back(perms[i]);
    }
    for(int i = 0; i < PERMS_KMERES; i++)
        permutationsMap[perms[i]] = i+1;
    /*
     * For k = 3 and |alphabet| = 4
     * we need we need 4**3 combinations size 3
     * 64 combinations size 3+1 bytes (end of string)
     * 192 bytes
     * */
    // We need to copy permutations to device constant memory
    // 65536 max constant memory
    if (VERBOSE){
        std::cout << "K = " << K << std::endl;
        std::cout << PERMS_KMERES * sizeof(char) * (K+1) << "/65536 bytes allocated" << std::endl;
        std::cout << "Máximum " << MAX_WORDS << " words in constant memory" << std::endl;
        std::cout << "(words size "<< K+1 << " bytes)";
        
    }




    // absolute path of the input data
    importSeqs(file);
    //importSeqsNoNL(file);
    resultsArraySize = numberOfSequenses*(numberOfSequenses+1) / 2 - numberOfSequenses;
    std::cout << "Size all seqs:" << size_all_seqs << std::endl;
    std::cout << seqs.size() << " sequences read ." << std::endl;

    doSequentialKmereDistance();
    printf("\n\aParallel:\n");
    // Device allocation
    doParallelKmereDistance();
    return 0;
}

void doSequentialKmereDistance(){
    // results files
    FILE *f_seq_res = fopen("/home/acervantes/kmerDist/sequential_results.csv", "w");
    //distancesSequential = (float**) malloc(sizeof(float*) * numberOfSequenses);
    //distancesParallel   = (float**) malloc(sizeof(float*) * numberOfSequenses);
    distancesSequential = (float*) calloc(resultsArraySize, sizeof(float));

    //    for (int i = 0; i < numberOfSequenses ; i++){
    //        for (int j = 0; j < numberOfSequenses ; j++) {
    //            distancesSequential[i][j] = -1;
    //        }
    //    }
    clock_t start_ser = clock();
    sequentialKmerCount2(seqs, permutationsList, 3);
    clock_t end_ser = clock();
    double serialTimer = 0;
    serialTimer = double (end_ser-start_ser) / double(CLOCKS_PER_SEC);
    cout << "Elapsed time serial: " << serialTimer << "[s]" << endl;
    if(PRINT_ANSWERS)
    for (long i = 0; i < resultsArraySize; i++){
        printf("%f\n", distancesSequential[i]);
    }

    if(PRINT_ANSWERS_FILE)
    for (long i = 0; i < resultsArraySize; i++){
        fprintf(f_seq_res,"%f\n", distancesSequential[i]);
    }
    /*for (long i = numberOfSequenses - 1, idx = 0; i > 0 ; i--){
        for (long j = 0; j < i ; j++, idx++) {
            fprintf(f_seq_res,"%f\t",distancesSequential[idx]);
            printf("%f(%ld)\t",distancesSequential[idx], idx);
            //distancesParallel[i][j] = 0;
        }
        fprintf(f_seq_res,"\n");
        printf("\n");
    }*/
    fclose(f_seq_res);
}

void doParallelKmereDistance(){
    FILE *f_res = fopen("/home/acervantes/kmerDist/parallel_results.csv", "w");
    hipError_t error;
    /**
     * Inicialización
     * */
    // Los índices de las entradas de las cadenas.
    int numIndexes = indexes_aux.size();
    error = hipMallocManaged(&indexes, numIndexes * sizeof(int));
    if (error){
        printf("Error malloc indexes: error #%d\n", error);
        exit(1);
    }
    for (int i = 0; i < indexes_aux.size(); i++){
        indexes[i] = indexes_aux[i];
    }

    // Suma de cada kmero de cada entrada.
    int numSumResults = numberOfSequenses*PERMS_KMERES; // sequences x 4**3
    int sumsSize = sizeof(int)*numSumResults;
    error = hipMallocManaged(&sums, sumsSize);
    if (error){
        printf("Error malloc indexes: error #%d\n", error);
        exit(1);
    }
    for (int i = 0; i < numSumResults; i++){
        sums[i] = 0;
    }


    /**
     * Mins: será una estructura de datos para minimizar el uso de memoria.
     * En total, sería de n (n+1) / 2 donde n es el número de muestras.
     * Pero como tampoco es necesaria la matriz principal, se resta N elementos
     */
    minsSize = (long) ((long)numberOfSequenses*((long)numberOfSequenses+1) / 2) - numberOfSequenses;
    error = hipMallocManaged(&mins, minsSize*sizeof(float));
    if (error){
        printf("Error malloc mins: error #%d\n", error);
        exit(1);
    }
    printf("%d sequences founded.\n", numberOfSequenses);
    printf("Allocating %ld elements of distance results.\n", minsSize);
    for(int i = 0; i < minsSize; i++)
        mins[i] = 0;
    //int blocks = 10;
    //int threads = 64;
    hipEvent_t start;
    hipEvent_t globalStart;
    hipEvent_t stop;
    hipEvent_t globalStop;
    hipError_t err_;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventCreate(&globalStart);
    hipEventCreate(&globalStop);


    // Launch kernel
    //int smSize = 49152;
    //sumKmereCoincidences<<<blocks, threads, smSize>>>(d_data, d_indices, numberOfSequenses, d_sums);
    /**
     * Primera parte: se obtiene la matriz d_mins que contiene las distancias mínimas
     * de todos los kmeros de cada entrada.
     * Km1S1, Km1S2, Km1S3, ... , Km1Sn
     * Km2S1, Km2S2, Km2S3, ... , Km2Sn
     * Km3S1, Km3S2, Km3S3, ... , Km3Sn
     * .
     * .
     * .
     * Km64S, Km64S2, Km64S3, ... , Km64Sn
     * */
    hipEventRecord(start, nullptr);
    hipEventRecord(globalStart, nullptr);
    printf("Running %ld blocks and %ld threads\n", blocks, threads);
    /***
     *  Para evitar desborde de memoria compartida, se calendariza
     *  el ciclo para alcanzar a procesar todos los kmeros.
     *  Cada hilo analiza un kmero en memoria compartida.
     *
     */
    // TODO: asignación dinámica para valores mayores a K=6
    

    //for(int perm_offset = 0; perm_offset < PERMS_KMERES; perm_offset += MAX_WORDS){
    for(int perm_offset = 0; perm_offset < 1 ; perm_offset += MAX_WORDS){
        // Actualizamos los valores de las permutaciones de k-meros
        for(int i = 0; i < MAX_WORDS && i < PERMS_KMERES; i++){
            //std::cout << "Copying:" << perms[i] << std::endl;
            err_ = hipMemcpyToSymbol(HIP_SYMBOL(c_perms), perms[i+perm_offset], (K+1), i*(K+1));
            if(err_){
                std::cout << "Error i=" << i << " OFFSET: " << perm_offset << " error #" << err_ << std::endl;
                return;
            }
        }
        // TODO: Si el número de permutaciones (MAX_WORDS) excede el número de hilos (1024), calendarizar también
        sumKmereCoincidencesGlobalMemory<<<blockThread1, threadsStep1>>>
                (data, indexes, numberOfSequenses, sums, perm_offset);
        hipDeviceSynchronize();
        err_ = hipGetLastError();
        if (err_)
            printf("LastError sumCoincidences #%d\n", err_);
        else
            printf("Running ok\n");
        return;        
    }
    return;
    // sumKmereCoincidencesGlobalMemory<<<blockThread1, threadsStep1>>>(data, indexes, numberOfSequenses, sums);
    // hipDeviceSynchronize();
    // err_ = hipGetLastError();
    // if (err_)
    //     printf("LastError sumCoincidences #%d\n", err_);
    hipFree(data);
    hipEventRecord(stop, nullptr);
    hipEventSynchronize(stop);
    float parallelTimer = 0;
    hipEventElapsedTime(&parallelTimer, start, stop);
    cout<< "Elapsed parallel timer step 1: " << parallelTimer << " ms, " << parallelTimer / 1000 << " secs" <<endl;
    /*printf("Sums:\n");
    for(int j = 0, idx = 0; j < PERMS_KMERES; j++){
        printf("%d: ", j);
        for(int i = 0; i < numberOfSequenses; i++){
            printf("%d,\t", sums[idx++]);
        }
        printf("\n");
    }
    printf("\n");*/
    /**
     * Paso 2: calcular las distancias de todo vs todo.
     *      Para toda cadena i:
     *        - Obtener distancia desde i+1 hasta n.
     * Paso 3: Al calcular las distancias, aplicar la fórmula.
     * Versión 1: Reducción de operaciones por llamada al kernel.
     * Versión 2: utilizar una localización de hilos tal que se ejecute siempre lo mismo y
     *            no se desperdicie memoria
     *
     * */
    //minKmereDist<<<10, 1024>>>(d_sums,d_distances, numberOfSequenses, 0);


    //minKmeres<<<blocks, 64>>>(d_sums, d_mins, numberOfSequenses);
    // sin ejecutar kernel tarda aprox 344 ms
    // ejecutando kernel 374 ms
    hipEventRecord(start, nullptr);
    for(int i = 0; i < numberOfSequenses; i++){
        for(int perm_offset = 0; perm_offset < PERMS_KMERES; perm_offset += (int)MAX_SHARED_MEM / sizeof(int)){
            minKmeres2<<<blocks, threads>>>(sums, mins, numberOfSequenses, i, indexes, perm_offset);
            hipDeviceSynchronize();
            err_ = hipGetLastError();
            if (err_){
                printf("LastError kmere dist: %d iteration %d\n", err_, i);
                exit(1);
            }
        }
    }
    hipDeviceSynchronize();
    err_ = hipGetLastError();
    if (err_)
        printf("LastError kmere dist: %d\n", err_);
    hipEventRecord(stop, nullptr);
    hipEventSynchronize(stop);
    parallelTimer = 0;
    hipEventElapsedTime(&parallelTimer, start, stop);
    cout<< "Elapsed parallel step 2 timer: " << parallelTimer << " ms, " << parallelTimer / 1000 << " secs" <<endl;

    hipEventRecord(globalStop,0);
    hipEventSynchronize(globalStop);
    parallelTimer = 0;
    hipEventElapsedTime(&parallelTimer, globalStart, globalStop);
    cout<< "Total time elapsed parallel: " << parallelTimer << " ms, " << parallelTimer / 1000 << " secs" <<endl;
    if(PRINT_ANSWERS)
    for (long i = 0; i < minsSize; i++){
        printf("%f\n", mins[i]);
    }
    if(PRINT_ANSWERS_FILE)
    for (long i = 0; i < resultsArraySize; i++){
        fprintf(f_res,"%f\n", mins[i]);
    }
    /*for (long i = numberOfSequenses - 1, idx = 0; i > 0 ; i--, idx++){
        for (long j = 0; j < i ; j++) {
            printf("%f\t",mins[idx]);
        }
        printf("\n");
    }*/

    /*printf("SumaMins:\n");
    for(int i = 0; i < minsSize; i++){
        printf("%f\t", mins[i]);
    }
    printf("\n");*/
    //printMinDistances(h_mins, minsSize, numberOfSequenses);
    /* // Para comprobar que los índices están bien.
    for(int i = 0; i < numIndexes - 1; i++){
        std::cout << "idx: " << indexes[i] << "\tCadena " << i << " :" << all_seqs+indexes[i]+1 << std::endl;
    }*/

    //unsigned long int sizeDistances = numberOfSequenses*numberOfSequenses * sizeof(float);
    //hipMemcpy(h_distances, d_distances, sizeDistances, hipMemcpyDeviceToHost);
    /*
    printf("Sums:\n");
    for(int j = 0, idx = 0; j < 64; j++){
        printf("%d: ", j);
        for(int i = 0; i < numberOfSequenses; i++){
            printf("%d,\t", h_sums[idx++]);
        }
        printf("\n");
    }
    */

    free(distancesSequential);
    //free(distancesParallel);
    hipFree(distances);
    hipFree(indexes);
    hipFree(sums);
    hipFree(mins);

    return;

}

void importSeqsNoNL(string inputFile){
    int indexCounter = 0;
    ifstream input(inputFile);
    if (!input.good()) {
        std::cerr << "Error opening: " << inputFile << " . Check your file or pathh." << std::endl;
        exit(0);
    }
    string line;
    string acc = "";
    string globalAcc = "";
    bool newSeq = false;

    // Iterate over all secuences
    while (getline(input, line)) {
        // line may be empty so you *must* ignore blank lines
        // or you have a crash waiting to happen with line[0]
        if(line.empty()){
            continue;
        }
        //read the header of
        if (line[0] == '>') {
            // store id
            ids.push_back(line);
            newSeq = true;
            continue;
        }
        if (newSeq) {
            newSeq = false;
            acc = line;
            while (getline(input, line)) {
                if(line[0] == '>') newSeq = true;
                if(line.empty() || line[0] == 13 || line[0] == '>'){
                    acc += "|";
                    seqs.push_back(acc);
                    indexes_aux.push_back(indexCounter);
                    indexCounter += acc.size();
                    globalAcc += acc;
                    acc = "";
                    break;
                }
                acc += line;
                if(seqs.size() >= MAX_SEQS) break;
            }
            if (acc != ""){
                acc += "|";
                seqs.push_back(acc);
                indexes_aux.push_back(indexCounter);
                indexCounter += acc.size();
                globalAcc += acc;
                acc = "";
                indexes_aux.push_back(indexCounter);
                if(seqs.size() >= MAX_SEQS) break;
            }
        }
    }
    int last_index = indexes_aux.size();
    numberOfSequenses = seqs.size();
    size_all_seqs = globalAcc.size()*sizeof(char);
    hipError_t error;
    error = hipMallocManaged(&data, size_all_seqs);
    if (error){
        printf("Error #%d allocating device memory with data.", error);
        exit(1);
    }
    for(int i = 0; i < globalAcc.size(); i++){
        if (globalAcc[i] == '|'){
            data[i] = '\0';
            continue;
        }
        data[i] = globalAcc[i];
    }
    return;
}
void importSeqs(string inputFile){
    int indexCounter = 0;
    ifstream input(inputFile);
    if (!input.good()) {
        std::cerr << "Error opening: " << inputFile << " . Check your file or pathh." << std::endl;
        exit(0);
    }
    string line;
    string acc = "";
    string globalAcc = "";
    bool newSeq = false;

    // Iterate over all secuences
    while (getline(input, line)) {
        // line may be empty so you *must* ignore blank lines
        // or you have a crash waiting to happen with line[0]
        if(line.empty()){
            continue;
        }
        //read the header of
        if (line[0] == '>') {
            // store id
            ids.push_back(line);
            newSeq = true;
            continue;
        }
        if (newSeq) {
            newSeq = false;
            acc = line;
            while (getline(input, line)) {
                if(line.empty() || line[0] == 13){
                    acc += "|";
                    seqs.push_back(acc);
                    indexes_aux.push_back(indexCounter);
                    indexCounter += acc.size();
                    globalAcc += acc;
                    acc = "";
                    break;
                }
                acc += line;
                if(seqs.size() >= MAX_SEQS) break;
            }
            if (acc != ""){
                acc += "|";
                seqs.push_back(acc);
                indexes_aux.push_back(indexCounter);
                indexCounter += acc.size();
                globalAcc += acc;
                acc = "";
                indexes_aux.push_back(indexCounter);
                if(seqs.size() >= MAX_SEQS) break;
            }
        }
    }
    int last_index = indexes_aux.size();
    numberOfSequenses = seqs.size();
    size_all_seqs = globalAcc.size()*sizeof(char);
    hipError_t error;
    error = hipMallocManaged(&data, size_all_seqs);
    if (error){
        printf("Error #%d allocating device memory with data.", error);
        exit(1);
    }
    for(int i = 0; i < globalAcc.size(); i++){
        if (globalAcc[i] == '|'){
            data[i] = '\0';
            continue;
        }
        data[i] = globalAcc[i];
    }
    return;
}

/*Versión 2: tarda menos pero utiliza más memoria*/
void sequentialKmerCount2(vector<string> &seqs, vector<string> &permutations , int k){
    string mers[4] = {"A","C","G","T"};
    long numberOfSequences = seqs.size();
    // |kmers| is at most 4**k = 4**3 = 64
    int max_combinations = pow(4,k);
    float distance;
    long sum;
    long minimum;
    long minLength;
    long i,j,p;
    long aux;
    int **countKmeres = new int*[numberOfSequences];
    // Getting distance of each kmere of each sequence
    for(i =  0; i < numberOfSequences; i++){
        countKmeres[i] = new int[max_combinations+1];
        permutationsCountAll(seqs[i], countKmeres[i], max_combinations, k);
    }
    for(i =  0; i < numberOfSequences - 1; i++){
        for(j = i + 1; j < numberOfSequences; j++){
            minLength = min(seqs[i].size() - 1, seqs[j].size() - 1);
            sum = 0;
            minimum = -1;
            aux =  getIdxTriangularMatrixRowMajorSeq(i +1 ,  (j - i), numberOfSequences);
            for(p = 1; p <= max_combinations; p++){
                minimum = min(countKmeres[i][p], countKmeres[j][p]);
                sum += minimum;
            }
            distance = 1 - (float) sum / (minLength - k + 1);
            distancesSequential[aux] = distance;
            //printf("Distance #%ld\t%f (i=%d, j=%d)\n", aux, distance, i, j );
            // distancesSequential[j][i] = distance;
        }
    }
    return;
}



void permutationsCountAll(string sequence, int * countResults, int max_combinations, int k){
    int sequence_len = sequence.size();
    string current_kmere;
    for(int i = 0; i < max_combinations + 1; i++)
        countResults[i] = 0;
    for(int i = 0; i < sequence_len - k ; i++){
        current_kmere = sequence.substr(i,k);
        // tomamos el índice 0 como error en caso de encontrar algún caracter fuera del alfabeto válido de entrada.
        countResults[permutationsMap[current_kmere]]++;
    }
}


void getPermutations(char *str, char* permutations, int last, int index){
    string stri;

    int i, len = strlen(str);
    for ( i = 0; i < len; i++ ) {
        permutations[index] = str[i] ;
        if (index == last){
            stri = permutations;
            permutationsList.push_back(stri);
        }
        else
            getPermutations (str, permutations, last, index+1);
    }
}

void printSeqs(){
    cout<< "total number of seqs: " << seqs.size() << endl;
    for (int i = 0; i<seqs.size(); i++){
        cout << ">" <<  seqs[i] << endl;
    }
}

long getIdxTriangularMatrixRowMajorSeq(long i, long j, long n){
    return (n * (i - 1) - (((i - 2) * (i - 1)) / 2)) + (j - i);
}